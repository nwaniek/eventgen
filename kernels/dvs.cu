#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <utility>
#include "dvs.h"
#include "frame.h"


/*
 * struct EventBuffer - store a certain amount of events in the form of a buffer
 */
struct EventBuffer {
	int counter;
	dvs_event_t events[];
};

static size_t get_event_buffer_size(int N)
{
	return sizeof(int) + N * sizeof(dvs_event_t);
}


__global__
void dvs_sim(
	int width, int height,
	unsigned char *left, unsigned char *right,
	int thresh, EventBuffer *buffer, uint64_t t)
{
	// as event generation is sparse, we can simply use atomicAdd here
	// without too much time penalty

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x >= width || y >= height) return;
	int idx = y * width + x;

	int diff = (int)left[idx] - (int)right[idx];

	// __syncthreads();

	// on-event
	if (diff > thresh)
		buffer->events[atomicAdd(&buffer->counter, 1)] =
			{1u, (uint16_t)x, (uint16_t)(height - y), t};
	// off-event
	else if (diff < -thresh)
		buffer->events[atomicAdd(&buffer->counter, 1)] =
			{0u, (uint16_t)x, (uint16_t)(height - y), t};
}


void
print_cuda_info()
{
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	std::cout << "maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl
		  << "maxThreadsDim:      [" <<
			props.maxThreadsDim[0] << ", " <<
			props.maxThreadsDim[1] << ", " <<
			props.maxThreadsDim[2] << "]" << std::endl
		  << "maxGridSize:        [" <<
			props.maxGridSize[0] << ", " <<
			props.maxGridSize[1] << ", " <<
			props.maxGridSize[2] << "]" << std::endl
		  ;
}


std::vector<dvs_event_t>
process_files(config_t &config, std::vector<std::string> &files)
{
	Frame *left = new Frame();
	left->load_from_file(files[0]);
	left->toGPU();
	int N = left->x * left->y;

	// allocate memory to store the events both on host and device
	size_t bufsize = get_event_buffer_size(N);
	EventBuffer *buf_a = (EventBuffer*)malloc(bufsize);
	EventBuffer *buf_b = (EventBuffer*)malloc(bufsize);
	memset(buf_a, 0, bufsize);
	memset(buf_b, 0, bufsize);

	EventBuffer *dev_buf_a;
	EventBuffer *dev_buf_b;
	hipMalloc((void**)&dev_buf_a, bufsize);
	hipMalloc((void**)&dev_buf_b, bufsize);

	// storage for the result
	std::vector<dvs_event_t> result;

	int64_t t = config.start_t;
	for (size_t i = 1; i < files.size(); i++) {
		Frame *right = new Frame();
		right->load_from_file(files[i]);

		// synchronization point: load data to/from GPU. blocking calls
		right->toGPU();
		hipMemcpy(dev_buf_a, buf_a, bufsize, hipMemcpyHostToDevice);
		hipMemcpy(buf_b, dev_buf_b, bufsize, hipMemcpyDeviceToHost);

		// call the CUDA kernel
		dim3 threadsPerBlock(32, 8);
		dim3 numBlocks(left->x / threadsPerBlock.x, left->y / threadsPerBlock.y);
		dvs_sim<<<numBlocks, threadsPerBlock>>>(
				left->x, left->y,
				left->dev_data, right->dev_data,
				config.thresh, dev_buf_a, t);

		// copy the events to the result vector
		for (int i = 0; i < buf_b->counter; i++)
			result.push_back({
					buf_b->events[i].polarity,
					buf_b->events[i].x,
					buf_b->events[i].y,
					buf_b->events[i].t});

		// reset the event buffer counter
		memset(buf_b, 0, bufsize);

		// wait for the device to finish
		hipDeviceSynchronize();

		// swap pointers
		std::swap(dev_buf_a, dev_buf_b);
		std::swap(buf_a, buf_b);
		std::swap(left, right);

		// remove "right" (which is the old left) as we don't need it
		// anymore
		delete right;
		t += config.delta_t;
	}

	delete left;
	hipFree(dev_buf_b);
	hipFree(dev_buf_a);
	free(buf_b);
	free(buf_a);

	return std::move(result);
}
